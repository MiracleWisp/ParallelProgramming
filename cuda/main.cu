#include "hip/hip_runtime.h"
#include <cstdio>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <chrono>

#define CUDA_RANGE 1000
#define BLOCK_SIZE 512
#define WARP_SIZE 32

struct point {
    float x;
    float y;
};

const int THREADS = 1 << 20;
const int THREADS_PER_BLOCK = 512;


__device__ float func(float x) {
    return cosf(3.0f * powf(x, 4.0f)) * sinf(5.0f * powf(x, 2.0f)) * powf(sinf(5.0f * x), 2.0f);

}

__device__ point max(point a, point b) {
    if (a.y > b.y) {
        return a;
    }
    return b;
}

__global__ void kernel(point *outData) {

    __shared__ point data[BLOCK_SIZE];
    int thread_id = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float part = (float) CUDA_RANGE / (float) (blockDim.x * gridDim.x);
    float end = part * (float) (idx + 1);
    float x = part * (float) idx;

    point result{
            x,
            func(x)
    };

    while (x <= end) {
        float y = func(x);
        if (y > result.y) {
            result.y = y;
            result.x = x;
        }
        x += 1E-4f;
    }

    data[thread_id] = result;
    __syncthreads();

    for (int s = blockDim.x / 2; s > WARP_SIZE; s >>= 1) {
        if (thread_id < s)
            data[thread_id] = max(data[thread_id], data[thread_id + s]);
        __syncthreads();
    }

    if (thread_id < WARP_SIZE) {
        data[thread_id] = max(data[thread_id], data[thread_id + 32]);
        data[thread_id] = max(data[thread_id], data[thread_id + 16]);
        data[thread_id] = max(data[thread_id], data[thread_id + 8]);
        data[thread_id] = max(data[thread_id], data[thread_id + 4]);
        data[thread_id] = max(data[thread_id], data[thread_id + 2]);
        data[thread_id] = max(data[thread_id], data[thread_id + 1]);
    }

    if (thread_id == 0)
        outData[blockIdx.x] = data[0];
}

int main() {
    auto start = std::chrono::steady_clock::now();
    thrust::host_vector<point> H(THREADS / THREADS_PER_BLOCK);
    thrust::device_vector<point> D(THREADS / THREADS_PER_BLOCK);
    point *ptr = thrust::raw_pointer_cast(&D[0]);
    kernel<<<THREADS / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(ptr);
    thrust::copy(D.begin(), D.end(), H.begin());

    point res{
            -1,
            -std::numeric_limits<float>::infinity()
    };

    for (int i = 0; i < H.size(); i++) {
        if (H[i].y > res.y) {
            res = H[i];
        }
    }
    auto end = std::chrono::steady_clock::now();
    std::cout << "X: " << res.x << " Y: " << res.y << std::endl;
    std::cout << "Duration: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "ms"
              << std::endl;

    return EXIT_SUCCESS;
}
